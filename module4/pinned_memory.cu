
#include <hip/hip_runtime.h>
#include <stdio.h>

//From https://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char** argv)
{
  int shift = 20;
  int iterations = 10;
  if (argc > 1) {
    shift = atoi(argv[1]);
  }
  if (argc > 2) {
    iterations = atoi(argv[2]);
  }

  int N = 1<<shift;
  int bytes = N*sizeof(float);
  float *x, *y, *d_x, *d_y;

  hipHostMalloc((void**)&x, bytes, hipHostMallocDefault);
  hipHostMalloc((void**)&y, bytes, hipHostMallocDefault);

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  for (int i = 0; i < iterations; ++i) {
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  }

  //float maxError = 0.0f;
  //for (int i = 0; i < N; i++){
    //maxError = max(maxError, abs(y[i]-4.0f));
    //printf("y[%d]=%f\n",i,y[i]);
  //}
  //printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  hipHostFree(x);
  hipHostFree(y);
}
